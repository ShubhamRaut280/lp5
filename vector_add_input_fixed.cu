
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n;
    scanf("%d", &n);  // Number of elements

    int *a = (int*)malloc(n * sizeof(int));
    int *b = (int*)malloc(n * sizeof(int));
    int *c = (int*)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        scanf("%d", &a[i]);
    }

    for (int i = 0; i < n; i++) {
        scanf("%d", &b[i]);
    }

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();  // Ensure GPU finishes before copying results

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);
    return 0;
}
